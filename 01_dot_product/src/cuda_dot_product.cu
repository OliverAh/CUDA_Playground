// cuda_dot_product.cu
#include <hip/hip_runtime.h>
//#include <stdio.h>

__global__ void dot_product_kernel(const float* a, const float* b, float* result, int n) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    float sum = 0.0;

    // Use a for loop to accumulate the dot product
    for (int i = index; i < n; i += blockDim.x * gridDim.x) {
        sum += a[i] * b[i];
    }

    atomicAdd(result, sum);
}

// Ensure C linkage for the following functions
extern "C" __declspec(dllexport) void dot_product(const float* a, const float* b, float* result, int n) {
    // Function implementation...
    // Allocate device memory
    float* d_a;
    float* d_b;
    float* d_result;

    hipMalloc((void**)& d_a, n * sizeof(float));  // Corrected line
    hipMalloc((void**)& d_b, n * sizeof(float));
    hipMalloc((void**)& d_result, sizeof(float));

    // Copy data to device
    hipMemcpy(d_a, a, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, n * sizeof(float), hipMemcpyHostToDevice);

    // Initialize result to 0 on the device
    float zero = 0.0f;
    hipMemcpy(d_result, &zero, sizeof(float), hipMemcpyHostToDevice);

    // Launch kernel with an appropriate number of blocks and threads
    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;

    dot_product_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_result, n);

   // Copy result back to host
   hipMemcpy(result, d_result, sizeof(float), hipMemcpyDeviceToHost);

   // Free device memory
   hipFree(d_a);
   hipFree(d_b);
   hipFree(d_result);

} // End of extern "C"